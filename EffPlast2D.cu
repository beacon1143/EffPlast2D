#include "hip/hip_runtime.h"
#include "EffPlast2D.h"

__global__ void ComputeDisp(double* Ux, double* Uy, double* Vx, double* Vy,
    const double* const P,
    const double* const tauXX, const double* const tauYY, const double* const tauXY,
    const double* const pa,
    const long int nX, const long int nY) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    const double dX = pa[0], dY = pa[1];
    const double dT = pa[2];
    const double rho = pa[5];
    const double dampX = pa[6], dampY = pa[7];

    // motion equation
    if (i > 0 && i < nX && j > 0 && j < nY - 1) {
        Vx[j * (nX + 1) + i] = Vx[j * (nX + 1) + i] * (1.0 - dT * dampX) + (dT / rho) * ((
            -P[j * nX + i] + P[j * nX + i - 1] + tauXX[j * nX + i] - tauXX[j * nX + i - 1]
            ) / dX + (
                tauXY[j * (nX - 1) + i - 1] - tauXY[(j - 1) * (nX - 1) + i - 1]
                ) / dY);
    }
    if (i > 0 && i < nX - 1 && j > 0 && j < nY) {
        Vy[j * nX + i] = Vy[j * nX + i] * (1.0 - dT * dampY) + (dT / rho) * ((
            -P[j * nX + i] + P[(j - 1) * nX + i] + tauYY[j * nX + i] - tauYY[(j - 1) * nX + i]
            ) / dY + (
                tauXY[(j - 1) * (nX - 1) + i] - tauXY[(j - 1) * (nX - 1) + i - 1]
                ) / dX);
    }

    Ux[j * (nX + 1) + i] = Ux[j * (nX + 1) + i] + Vx[j * (nX + 1) + i] * dT;
    Uy[j * nX + i] = Uy[j * nX + i] + Vy[j * nX + i] * dT;
}

__global__ void ComputeStress(const double* const Ux, const double* const Uy,
    const double* const K, const double* const G,
    const double* const P0, double* P,
    double* tauXX, double* tauYY, double* tauXY,
    const double* const pa,
    const long int nX, const long int nY) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    const double dX = pa[0], dY = pa[1];
    // const double dT = pa[2];
    const double rad = pa[9];
    const double N = pa[10];

    // constitutive equation - Hooke's law
    P[j * nX + i] = P0[j * nX + i] - K[j * nX + i] * (
        (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY    // divU
        );

    /*P[j * nX + i] = P[j * nX + i] - G[j * nX + i] * ( // incompressibility
                    (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY    // divU
                    ) * dT / nX;*/

    tauXX[j * nX + i] = 2.0 * G[j * nX + i] * (
        (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX -    // dUx/dx
        ((Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY) / 3.0    // divU / 3.0
        );
    tauYY[j * nX + i] = 2.0 * G[j * nX + i] * (
        (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY -    // dUy/dy
        ((Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY) / 3.0    // divU / 3.0
        );

    if (i < nX - 1 && j < nY - 1) {
        tauXY[j * (nX - 1) + i] = 0.25 * (G[j * nX + i] + G[j * nX + i + 1] + G[(j + 1) * nX + i] + G[(j + 1) * nX + i + 1]) * (
            (Ux[(j + 1) * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i + 1]) / dY + (Uy[(j + 1) * nX + i + 1] - Uy[(j + 1) * nX + i]) / dX    // dUx/dy + dUy/dx
            );
    }

    for (int k = 0; k < N; k++) {
        for (int l = 0; l < N; l++) {
            if (sqrt((-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) *
                (-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) +
                (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l) *
                (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l)) < rad) {
                P[j * nX + i] = 0.0;
                tauXX[j * nX + i] = 0.0;
                tauYY[j * nX + i] = 0.0;
            }

            if (i < nX - 1 && j < nY - 1) {
                if (sqrt((-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) *
                    (-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) +
                    (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l) *
                    (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l)) < rad) {
                    tauXY[j * (nX - 1) + i] = 0.0;
                }
            }
        }
    }
}

__global__ void ComputeJ2(double* tauXX, double* tauYY, double* tauXY, 
    double* const tauXYav, 
    double* const J2, double* const J2XY,
    const long int nX, const long int nY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // tauXY for plasticity
    if (i > 0 && i < nX - 1 &&
        j > 0 && j < nY - 1) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 1) * (nX - 1) + i - 1] + tauXY[(j - 1) * (nX - 1) + i] + tauXY[j * (nX - 1) + i - 1] + tauXY[j * (nX - 1) + i]);
    }
    else if (i == 0 && j > 0 && j < nY - 1) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 1) * (nX - 1) + i] + tauXY[(j - 1) * (nX - 1) + i + 1] + tauXY[j * (nX - 1) + i] + tauXY[j * (nX - 1) + i + 1]);
    }
    else if (i == nX - 1 && j > 0 && j < nY - 1) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 1) * (nX - 1) + i - 2] + tauXY[(j - 1) * (nX - 1) + i - 1] + tauXY[j * (nX - 1) + i - 2] + tauXY[j * (nX - 1) + i - 1]);
    }
    else if (i > 0 && i < nX - 1 && j == 0) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[j * (nX - 1) + i - 1] + tauXY[j * (nX - 1) + i] + tauXY[(j + 1) * (nX - 1) + i - 1] + tauXY[(j + 1) * (nX - 1) + i]);
    }
    else if (i > 0 && i < nX - 1 && j == nY - 1) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 2) * (nX - 1) + i - 1] + tauXY[(j - 2) * (nX - 1) + i] + tauXY[(j - 1) * (nX - 1) + i - 1] + tauXY[(j - 1) * (nX - 1) + i]);
    }
    else if (i == 0 && j == 0) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[j * (nX - 1) + i] + tauXY[j * (nX - 1) + i + 1] + tauXY[(j + 1) * (nX - 1) + i] + tauXY[(j + 1) * (nX - 1) + i + 1]);
    }
    else if (i == 0 && j == nY - 1) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 2) * (nX - 1) + i] + tauXY[(j - 2) * (nX - 1) + i + 1] + tauXY[(j - 1) * (nX - 1) + i] + tauXY[(j - 1) * (nX - 1) + i + 1]);
    }
    else if (i == nX - 1 && j == 0) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[j * (nX - 1) + i - 2] + tauXY[j * (nX - 1) + i - 1] + tauXY[(j + 1) * (nX - 1) + i - 2] + tauXY[(j + 1) * (nX - 1) + i - 1]);
    }
    else if (i == nX - 1 && j == nY - 1) {
        tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 2) * (nX - 1) + i - 2] + tauXY[(j - 2) * (nX - 1) + i - 1] + tauXY[(j - 1) * (nX - 1) + i - 2] + tauXY[(j - 1) * (nX - 1) + i - 1]);
    }

    J2[j * nX + i] = sqrt(tauXX[j * nX + i] * tauXX[j * nX + i] + tauYY[j * nX + i] * tauYY[j * nX + i] + 2.0 * tauXYav[j * nX + i] * tauXYav[j * nX + i]);
    if (i < nX - 1 && j < nY - 1) {
        J2XY[j * (nX - 1) + i] = sqrt(
            0.0625 * (tauXX[j * nX + i] + tauXX[j * nX + i + 1] + tauXX[(j + 1) * nX + i] + tauXX[(j + 1) * nX + i + 1]) * (tauXX[j * nX + i] + tauXX[j * nX + i + 1] + tauXX[(j + 1) * nX + i] + tauXX[(j + 1) * nX + i + 1]) +
            0.0625 * (tauYY[j * nX + i] + tauYY[j * nX + i + 1] + tauYY[(j + 1) * nX + i] + tauYY[(j + 1) * nX + i + 1]) * (tauYY[j * nX + i] + tauYY[j * nX + i + 1] + tauYY[(j + 1) * nX + i] + tauYY[(j + 1) * nX + i + 1]) +
            2.0 * tauXY[j * (nX - 1) + i] * tauXY[j * (nX - 1) + i]
        );
    }

}

__global__ void ComputePlasticity(double* tauXX, double* tauYY, double* tauXY,
    double* const tauXYav,
    double* const J2, double* const J2XY,
    const double* const pa,
    const long int nX, const long int nY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //const double dX = pa[0], dY = pa[1];
    const double coh = pa[8];
    //const double rad = pa[9];

    /*if (sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) < rad ) {
      tauXYav[j * nX + i] = 0.0;
    }*/

    // plasticity
    if (J2[j * nX + i] > coh) {
        tauXX[j * nX + i] *= coh / J2[j * nX + i];
        tauYY[j * nX + i] *= coh / J2[j * nX + i];
        tauXYav[j * nX + i] *= coh / J2[j * nX + i];
        J2[j * nX + i] = sqrt(tauXX[j * nX + i] * tauXX[j * nX + i] + tauYY[j * nX + i] * tauYY[j * nX + i] + 2.0 * tauXYav[j * nX + i] * tauXYav[j * nX + i]);
    }

    if (i < nX - 1 && j < nY - 1) {
        if (J2XY[j * (nX - 1) + i] > coh) {
            tauXY[j * (nX - 1) + i] *= coh / J2XY[j * (nX - 1) + i];
        }
    }
}

std::array<std::vector<std::array<double, 3>>, NL> EffPlast2D::ComputeSigma(
	const double initLoadValue, 
	const double loadValue, 
    const unsigned int nTimeSteps, 
	const std::array<double, 3>& loadType
)
{    
    log_file << "init load: (" << initLoadValue * loadType[0] << ", " << initLoadValue * loadType[1] << ", " << initLoadValue * loadType[2] << ")\n" 
             << "   + load: (" << loadValue * loadType[0] << ", " << loadValue * loadType[1] << ", " << loadValue * loadType[2] << ") x" << (nTimeSteps - 1) << '\n';
    std::cout << "init load: (" << initLoadValue * loadType[0] << ", " << initLoadValue * loadType[1] << ", " << initLoadValue * loadType[2] << ")\n" 
              << "   + load: (" << loadValue * loadType[0] << ", " << loadValue * loadType[1] << ", " << loadValue * loadType[2] << ") x" << (nTimeSteps - 1) << '\n';

    const double incPercent = 0.005;
    const double incLoad =  0.5 * (loadValue * loadType[0] + loadValue * loadType[1]) * incPercent;

    std::array<std::vector<std::array<double, 3>>, NL> Sigma;
    std::array<std::vector<double>, NL> deltaP;
    std::array<std::vector<double>, NL> dPhi;

    for (int nload = 0; nload < NL; nload++)
    {
        Sigma[nload].resize(nTimeSteps);
        deltaP[nload].resize(nTimeSteps);
        dPhi[nload].resize(nTimeSteps);

        double dUxdx = initLoadValue * loadType[0];
        double dUydy = initLoadValue * loadType[1];
        double dUxdy = initLoadValue * loadType[2];

        memset(Ux_cpu, 0, (nX + 1) * nY * sizeof(double));
        memset(Uy_cpu, 0, nX * (nY + 1) * sizeof(double));

        /* ACTION LOOP */
        for (int it = 0; it < nTimeSteps; it++) {
            log_file << "\n\nload step " << (it + 1) << '\n';
            std::cout << "\n\nload step " << (it + 1) << '\n';

            if (it > 0)
            {
                dUxdx = loadValue * loadType[0];
                dUydy = loadValue * loadType[1];
                dUxdy = loadValue * loadType[2];

                gpuErrchk(hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost));
                gpuErrchk(hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost));
            }

            for (int i = 0; i < nX + 1; i++) {
                for (int j = 0; j < nY; j++) {
                    Ux_cpu[j * (nX + 1) + i] += ((-0.5 * dX * nX + dX * i) * (dUxdx) + (-0.5 * dY * (nY - 1) + dY * j) * dUxdy) * (1.0 + nload * incPercent);
                }
            }
            gpuErrchk(hipMemcpy(Ux_cuda, Ux_cpu, (nX + 1) * nY * sizeof(double), hipMemcpyHostToDevice));

            for (int i = 0; i < nX; i++) {
                for (int j = 0; j < nY + 1; j++) {
                    Uy_cpu[j * nX + i] += ((-0.5 * dY * nY + dY * j) * (dUydy)) * (1.0 + nload * incPercent);
                }
            }
            gpuErrchk(hipMemcpy(Uy_cuda, Uy_cpu, nX * (nY + 1) * sizeof(double), hipMemcpyHostToDevice));

            double error = 0.0;

            /* ITERATION LOOP */
            for (int iter = 0; iter < NITER; iter++) {
                ComputeStress<<<grid, block>>>(Ux_cuda, Uy_cuda, K_cuda, G_cuda, P0_cuda, P_cuda, tauXX_cuda, tauYY_cuda, tauXY_cuda, /*tauXYav_cuda, J2_cuda, J2XY_cuda,*/ pa_cuda, nX, nY);
                gpuErrchk(hipDeviceSynchronize());
                ComputeJ2<<<grid, block>>>(tauXX_cuda, tauYY_cuda, tauXY_cuda, tauXYav_cuda, J2_cuda, J2XY_cuda, nX, nY);
                gpuErrchk(hipDeviceSynchronize());
                ComputePlasticity<<<grid, block>>>(tauXX_cuda, tauYY_cuda, tauXY_cuda, tauXYav_cuda, J2_cuda, J2XY_cuda, pa_cuda, nX, nY);
                gpuErrchk(hipDeviceSynchronize());
                ComputeDisp<<<grid, block>>>(Ux_cuda, Uy_cuda, Vx_cuda, Vy_cuda, P_cuda, tauXX_cuda, tauYY_cuda, tauXY_cuda, pa_cuda, nX, nY);
                gpuErrchk(hipDeviceSynchronize());

                if ((iter + 1) % output_step == 0) {
                    gpuErrchk(hipMemcpy(Vx_cpu, Vx_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost));
                    gpuErrchk(hipMemcpy(Vy_cpu, Vy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost));

                    error = (FindMaxAbs(Vx_cpu, (nX + 1) * nY) / (dX * (nX - 1)) + FindMaxAbs(Vy_cpu, nX * (nY + 1)) / (dY * (nY - 1))) * dT /
                        (std::abs(loadValue) * std::max(std::max(std::abs(loadType[0]), std::abs(loadType[1])), std::abs(loadType[2])));

                    std::cout << "Iteration " << iter + 1 << ": Error is " << error << '\n';
                    log_file << "Iteration " << iter + 1 << ": Error is " << error << '\n';

                    if (error < EITER) {
                        std::cout << "Number of iterations is " << iter + 1 << '\n';
                        log_file << "Number of iterations is " << iter + 1 << '\n';
                        break;
                    }
                    else if (iter == NITER - 1) {
                        std::cout << "WARNING: Maximum number of iterations reached!\nError is " << error << '\n';
                        log_file << "WARNING: Maximum number of iterations reached!\nError is " << error << '\n';
                    }
                }
            }
            /* AVERAGING */
            gpuErrchk(hipMemcpy(P_cpu, P_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(tauXX_cpu, tauXX_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(tauYY_cpu, tauYY_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(tauXY_cpu, tauXY_cuda, (nX - 1) * (nY - 1) * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(J2_cpu, J2_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost));

            /*Sigma[nload][it] = {0.0, 0.0, 0.0};
            for (int i = 0; i < nX; i++) {
              for (int j = 0; j < nY; j++) {
                Sigma[nload][it][0] += tauXX_cpu[j * nX + i] - P_cpu[j * nX + i];
                Sigma[nload][it][1] += tauYY_cpu[j * nX + i] - P_cpu[j * nX + i];
              }
            }
            Sigma[nload][it][0] /= nX * nY;
            Sigma[nload][it][1] /= nX * nY;

            for (int i = 0; i < nX - 1; i++) {
              for (int j = 0; j < nY - 1; j++) {
                Sigma[nload][it][2] += tauXY_cpu[j * (nX - 1) + i];
              }
            }
            Sigma[nload][it][2] /= (nX - 1) * (nY - 1);*/

            // -P_eff
            for (int i = 0; i < nX; i++) {
                for (int j = 0; j < nY; j++) {
                    if (sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) >= rad) {
                        Sigma[nload][it][0] += -P_cpu[j * nX + i];
                    }
                    else {
                        // std::cout << "In the hole!\n";
                        // log_file << "In the hole!\n";
                    }
                }
            }
            Sigma[nload][it][0] /= nX * nY;

            // Tau_eff
            for (int i = 0; i < nX; i++) {
                for (int j = 0; j < nY; j++) {
                    if (sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) >= rad) {
                        Sigma[nload][it][1] += tauXX_cpu[j * nX + i];
                        Sigma[nload][it][2] += tauYY_cpu[j * nX + i];
                    }
                }
            }
            Sigma[nload][it][1] /= nX * nY;
            Sigma[nload][it][2] /= nX * nY;

            // std::cout << Sigma[nload][it][0] / loadValue << '\t' << Sigma[nload][it][1] / loadValue << '\t' << Sigma[nload][it][2] / loadValue << '\n';
            // log_file << Sigma[nload][it][0] / loadValue << '\t' << Sigma[nload][it][1] / loadValue << '\t' << Sigma[nload][it][2] / loadValue << '\n';

            /* ANALYTIC SOLUTION FOR EFFECTIVE PROPERTIES */
            deltaP[nload][it] = GetDeltaP_approx(loadValue * loadType[0], loadValue * loadType[1]); // GetDeltaP_honest();
            std::cout << "deltaP = " << deltaP[nload][it] << '\n';
            log_file << "deltaP = " << deltaP[nload][it] << '\n';
            //const double deltaP = GetDeltaP_approx(loadValue * loadType[0], loadValue * loadType[1]);
            const double tauInfty_approx = GetTauInfty_approx(loadValue * loadType[0], loadValue * loadType[1]); // GetTauInfty_honest();

            int holeX = static_cast<int>((nX + 1) * 2 * rad / nX / dX);    // approx X-axis index of hole boundary
            std::vector<double> dispX((nX + 1) / 2);
            for (int i = (nX + 1) / 2 - holeX - 1; i < (nX + 1) / 2; i++) {
                dispX[i] = Ux_cpu[(nY / 2) * (nX + 1) + i];
            }

            int holeY = static_cast<int>((nY + 1) * 2 * rad / nY / dY);    // approx Y-axis index of hole boundary
            std::vector<double> dispY((nY + 1) / 2);
            for (int j = (nY + 1) / 2 - holeY - 1; j < (nY + 1) / 2; j++) {
                dispY[j] = Uy_cpu[j * nX + nX / 2];
            }

            /*std::vector<double> dispXwrong((nY + 1) / 2);
            for (int j = nY / 2 - holeY - 2; j < nY / 2; j++) {
              dispXwrong[j] = Ux_cpu[j * nX + nX / 2];
            }*/

            /*const double dR = FindMaxAbs(Ux_cpu, (nX + 1) * nY);
            std::cout << "dR = " << dR << '\n';
            log_file << "dR = " << dR << '\n';*/
            const double dRx = -FindMaxAbs(dispX);
            //std::cout << "dRx = " << dRx << '\n';
            log_file << "dRx = " << dRx << '\n';
            const double dRy = -FindMaxAbs(dispY);
            //std::cout << "dRy = " << dRy << '\n';
            log_file << "dRy = " << dRy << '\n';
            /*const double dRxWrong = -FindMaxAbs(dispXwrong);
            std::cout << "dRxWrong = " << dRxWrong << '\n';*/
            const double Phi0 = 3.1415926 * rad * rad / (dX * (nX - 1) * dY * (nY - 1));
            const double Phi = 3.1415926 * (rad + dRx) * (rad + dRy) / (dX * (nX - 1) * dY * (nY - 1) * (1 + loadValue * loadType[0]) * (1 + loadValue * loadType[1]));
            dPhi[nload][it] = 3.1415926 * (std::abs((rad + dRx) * (rad + dRy) - rad * rad)) / (dX * (nX - 1) * dY * (nY - 1));
            //std::cout << "dPhi = " << dPhi[nload][it] << '\n';
            log_file << "dPhi = " << dPhi[nload][it] << '\n';

            const double KeffPhi = deltaP[nload][it] / dPhi[nload][it];
            //const double KeffPhi = deltaP_honest / dPhi;

            //std::cout << "deltaP_honest = " << deltaP_honest << '\n';
            //log_file << "deltaP_honest = " << deltaP_honest << '\n';
            std::cout << "deltaP / Y = " << deltaP[nload][it] / Y << '\n';
            log_file << "deltaP / Y = " << deltaP[nload][it] / Y << '\n';
            //std::cout << "tauInfty / Y = " << tauInfty_approx / Y << '\n';
            log_file << "tauInfty / Y = " << tauInfty_approx / Y << '\n';
            //std::cout << "KeffPhi = " << KeffPhi << '\n';
            log_file << "KeffPhi = " << KeffPhi << '\n';

            const double phi = 3.1415926 * rad * rad / (dX * (nX - 1) * dY * (nY - 1));
            const double KexactElast = G0 / phi;
            const double KexactPlast = G0 / (phi - dPhi[nload][it]) / exp(std::abs(deltaP[nload][it]) / Y - 1.0) / // phi or phi - dPhi ?
                (1.0 + 5.0 * tauInfty_approx * tauInfty_approx / Y / Y);
            //const double KexactPlast = G0 / phi / exp(std::abs(deltaP_honest) / pa_cpu[8] - 1.0);
            //std::cout << "KexactElast = " << KexactElast << '\n';
            log_file << "KexactElast = " << KexactElast << '\n';
            std::cout << "KexactPlast = " << KexactPlast << '\n';
            log_file << "KexactPlast = " << KexactPlast << '\n';

            if (it + 1 == nTimeSteps && nload + 1 == NL)
                SaveAnStatic1D(deltaP[nload][it], tauInfty_approx);
        }
    }

    if (NL > 1)
    {
        const double KeffPhi = (deltaP[NL - 1][nTimeSteps - 1] - deltaP[NL - 2][nTimeSteps - 1]) / 
            (dPhi[NL - 1][nTimeSteps - 1] - dPhi[NL - 2][nTimeSteps - 1]);
        
        std::cout << "==============\n" << "KeffPhi = " << KeffPhi << '\n';
        log_file << "==============\n" << "KeffPhi = " << KeffPhi << '\n';
    }

    /* ANALYTIC 2D SOLUTION FOR STATICS */

    /* OUTPUT DATA WRITING */
    SaveMatrix(P_cpu, P_cuda, nX, nY, "Pc_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(tauXX_cpu, tauXX_cuda, nX, nY, "tauXXc_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(tauYY_cpu, tauYY_cuda, nX, nY, "tauYYc_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(tauXY_cpu, tauXY_cuda, nX - 1, nY - 1, "tauXYc_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(tauXYav_cpu, tauXYav_cuda, nX, nY, "tauXYavc_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(J2_cpu, J2_cuda, nX, nY, "J2c_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(Ux_cpu, Ux_cuda, nX + 1, nY, "Uxc_" + std::to_string(32 * NGRID) + "_.dat");
    SaveMatrix(Uy_cpu, Uy_cuda, nX, nY + 1, "Uyc_" + std::to_string(32 * NGRID) + "_.dat");

    //gpuErrchk(hipDeviceReset());
    return Sigma;
}

void EffPlast2D::ReadParams(const std::string& filename) {
    std::ifstream pa_fil(filename, std::ios_base::binary);
    if (!pa_fil) {
        std::cerr << "Error! Cannot open file pa.dat!\n";
        exit(1);
    }
    pa_fil.read((char*)pa_cpu, sizeof(double) * NPARS);
    gpuErrchk(hipMemcpy(pa_cuda, pa_cpu, NPARS * sizeof(double), hipMemcpyHostToDevice));
}

void EffPlast2D::SetMaterials() {
    //constexpr double K0 = 10.0;
    //constexpr double G0 = 0.01;

    for (int i = 0; i < nX; i++) {
        for (int j = 0; j < nY; j++) {
            K_cpu[j * nX + i] = K0;
            G_cpu[j * nX + i] = G0;
            double x = -0.5 * dX * (nX - 1) + dX * i;
            double y = -0.5 * dY * (nY - 1) + dY * j;
            double Lx = dX * (nX - 1);
            double Ly = dY * (nY - 1);
            for (int k = 0; k < N; k++) {
                for (int l = 0; l < N; l++) {
                    if (sqrt((x - 0.5 * Lx * (1.0 - 1.0 / N) + (Lx / N) * k) * (x - 0.5 * Lx * (1.0 - 1.0 / N) + (Lx / N) * k) +
                        (y - 0.5 * Ly * (1.0 - 1.0 / N) + (Ly / N) * l) * (y - 0.5 * Ly * (1.0 - 1.0 / N) + (Ly / N) * l)) < rad) {
                        K_cpu[j * nX + i] = 0.01 * K0;
                        G_cpu[j * nX + i] = 0.01 * G0;
                    }
                }
            }
        }
    }

    gpuErrchk(hipMemcpy(K_cuda, K_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(G_cuda, G_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice));
}

void EffPlast2D::SetInitPressure(const double coh) {
    const double P0 = 0.0; //1.0 * coh;

    for (int i = 0; i < nX; i++) {
        for (int j = 0; j < nY; j++) {
            P0_cpu[j * nX + i] = 0.0;
            if (sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) < rad) {
                P0_cpu[j * nX + i] = P0;
            }
        }
    }

    gpuErrchk(hipMemcpy(P0_cuda, P0_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice));
}

void EffPlast2D::SetMatrixZero(double** A_cpu, double** A_cuda, const int m, const int n) {
    *A_cpu = new double[m * n];
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            (*A_cpu)[j * m + i] = 0.0;
        }
    }
    gpuErrchk(hipMalloc(A_cuda, m * n * sizeof(double)));
    gpuErrchk(hipMemcpy(*A_cuda, *A_cpu, m * n * sizeof(double), hipMemcpyHostToDevice));
}

void EffPlast2D::SaveMatrix(double* const A_cpu, const double* const A_cuda, const int m, const int n, const std::string& filename) {
    gpuErrchk(hipMemcpy(A_cpu, A_cuda, m * n * sizeof(double), hipMemcpyDeviceToHost));
    std::ofstream A_filw(filename, std::ios_base::binary);
    A_filw.write((char*)A_cpu, sizeof(double) * m * n);
}

void EffPlast2D::SaveVector(double* const arr, const int size, const std::string& filename) {
    std::ofstream arr_filw(filename, std::ios_base::binary);
    arr_filw.write((char*)arr, sizeof(double) * size);
}

double EffPlast2D::FindMaxAbs(const double* const arr, const int size) {
    double max_el = 0.0;
    for (int i = 0; i < size; i++) {
        if (std::abs(arr[i]) > max_el) {
            max_el = std::abs(arr[i]);
        }
    }
    return max_el;
}

double EffPlast2D::FindMaxAbs(const std::vector<double>& vec) {
    double max_el = 0.0;
    for (auto i : vec) {
        if (std::abs(i) > max_el) {
            max_el = i;
        }
    }
    return max_el;
}

double EffPlast2D::GetDeltaP_honest() {
    double deltaP = 0.0, deltaPx = 0.0, deltaPy = 0.0;

    for (int i = 1; i < nX - 1; i++) {
        deltaPx += tauXX_cpu[0 * nX + i] - P_cpu[0 * nX + i];
        deltaPx += tauYY_cpu[0 * nX + i] - P_cpu[0 * nX + i];
        deltaPx += tauXX_cpu[(nY - 1) * nX + i] - P_cpu[(nY - 1) * nX + i];
        deltaPx += tauYY_cpu[(nY - 1) * nX + i] - P_cpu[(nY - 1) * nX + i];
    }
    deltaPx /= (nX - 2);

    for (int j = 1; j < nY - 1; j++) {
        deltaPy += tauXX_cpu[j * nX + 0] - P_cpu[j * nX + 0];
        deltaPy += tauYY_cpu[j * nX + 0] - P_cpu[j * nX + 0];
        deltaPy += tauXX_cpu[j * nX + nY - 1] - P_cpu[j * nX + nY - 1];
        deltaPy += tauYY_cpu[j * nX + nY - 1] - P_cpu[j * nX + nY - 1];
    }
    deltaPy /= (nY - 2);

    deltaP = -0.125 * (deltaPx + deltaPy);
    return deltaP;
}

double EffPlast2D::GetDeltaP_approx(const double Exx, const double Eyy) {
    double deltaP = 0.0;

    /*if (Exx < Eyy) {
        deltaP += tauXX_cpu[(nY / 2) * nX + 0] - P_cpu[(nY / 2) * nX + 0];
        deltaP += tauYY_cpu[(nY / 2) * nX + 0] - P_cpu[(nY / 2) * nX + 0];
        deltaP += tauXX_cpu[(nY / 2) * nX + nX - 1] - P_cpu[(nY / 2) * nX + nX - 1];
        deltaP += tauYY_cpu[(nY / 2) * nX + nX - 1] - P_cpu[(nY / 2) * nX + nX - 1];
    }
    else {
        deltaP += tauXX_cpu[0 * nX + nX / 2] - P_cpu[0 * nX + nX / 2];
        deltaP += tauYY_cpu[0 * nX + nX / 2] - P_cpu[0 * nX + nX / 2];
        deltaP += tauXX_cpu[(nY - 1) * nX + nX / 2] - P_cpu[(nY - 1) * nX + nX / 2];
        deltaP += tauYY_cpu[(nY - 1) * nX + nX / 2] - P_cpu[(nY - 1) * nX + nX / 2];
    }

    deltaP *= -0.25;*/

    deltaP += tauXX_cpu[(nY / 2) * nX + 0] - P_cpu[(nY / 2) * nX + 0];
    deltaP += tauYY_cpu[(nY / 2) * nX + 0] - P_cpu[(nY / 2) * nX + 0];
    deltaP += tauXX_cpu[(nY / 2) * nX + nX - 1] - P_cpu[(nY / 2) * nX + nX - 1];
    deltaP += tauYY_cpu[(nY / 2) * nX + nX - 1] - P_cpu[(nY / 2) * nX + nX - 1];
    deltaP += tauXX_cpu[0 * nX + nX / 2] - P_cpu[0 * nX + nX / 2];
    deltaP += tauYY_cpu[0 * nX + nX / 2] - P_cpu[0 * nX + nX / 2];
    deltaP += tauXX_cpu[(nY - 1) * nX + nX / 2] - P_cpu[(nY - 1) * nX + nX / 2];
    deltaP += tauYY_cpu[(nY - 1) * nX + nX / 2] - P_cpu[(nY - 1) * nX + nX / 2];

    deltaP *= -0.125;
    return deltaP;
}

double EffPlast2D::GetTauInfty_honest() {
    double tauInfty = 0.0, tauInftyx = 0.0, tauInftyy = 0.0;

    for (int i = 1; i < nX - 1; i++) {
        tauInftyx += tauXX_cpu[0 * nX + i] - tauYY_cpu[0 * nX + i];
        tauInftyx += tauXX_cpu[(nY - 1) * nX + i] - tauYY_cpu[(nY - 1) * nX + i];
    }
    tauInftyx /= (nX - 2);

    for (int j = 1; j < nY - 1; j++) {
        tauInftyy += tauXX_cpu[j * nX + 0] - tauYY_cpu[j * nX + 0];
        tauInftyy += tauXX_cpu[j * nX + nY - 1] - tauYY_cpu[j * nX + nY - 1];
    }
    tauInftyy /= (nY - 2);

    tauInfty = -0.125 * (tauInftyx + tauInftyy);
    return tauInfty;
}

double EffPlast2D::GetTauInfty_approx(const double Exx, const double Eyy) {
    double tauInfty = 0.0;

    /*if (Exx < Eyy) {
        tauInfty += tauYY_cpu[(nY / 2) * nX + 0] - tauXX_cpu[(nY / 2) * nX + 0];
        tauInfty += tauYY_cpu[(nY / 2) * nX + nX - 1] - tauXX_cpu[(nY / 2) * nX + nX - 1];
    }
    else {
        tauInfty += tauYY_cpu[0 * nX + nX / 2] - tauXX_cpu[0 * nX + nX / 2];
        tauInfty += tauYY_cpu[(nY - 1) * nX + nX / 2] - tauXX_cpu[(nY - 1) * nX + nX / 2];
    }

    tauInfty *= 0.25;*/

    tauInfty += tauYY_cpu[(nY / 2) * nX + 0] - tauXX_cpu[(nY / 2) * nX + 0];
    tauInfty += tauYY_cpu[(nY / 2) * nX + nX - 1] - tauXX_cpu[(nY / 2) * nX + nX - 1];
    tauInfty += tauYY_cpu[0 * nX + nX / 2] - tauXX_cpu[0 * nX + nX / 2];
    tauInfty += tauYY_cpu[(nY - 1) * nX + nX / 2] - tauXX_cpu[(nY - 1) * nX + nX / 2];

    tauInfty *= 0.125;

    return tauInfty;
}

void EffPlast2D::SaveAnStatic1D(const double deltaP, const double tauInfty) {
    /* ANALYTIC 1D SOLUTION FOR STATICS */
    const double Rmin = rad + 0.0 * dX;
    const double Rmax = 0.5 * dX * (nX - 1) - dX * 60.0;

    const double xi = (deltaP > 0.0) ? 1.0 : -1.0;
    const double kappa = tauInfty / Y * xi;
    const double c0 = rad * exp(abs(deltaP) / 2.0 / Y - 0.5);

    const double Rx = c0 * (1.0 - kappa);
    const double Ry = c0 * (1.0 + kappa);

    // double Rxnu = 0.5 * (nX - 1) * dX;
    // for (int i = 0; i < nX / 2; i++)
    //     if (J2_cpu[(nY / 2) * nX + i] <= (1.0 - 2.0 * std::numeric_limits<double>::epsilon()) * pa_cpu[8])
    //         Rxnu -= dX;
    //     else
    //         break;

    // double Rynu = 0.5 * (nY - 1) * dY;
    // for (int i = 0; i < nY / 2; i++)
    //     if (J2_cpu[i * nX + (nX / 2)] <= (1.0 - 2.0 * std::numeric_limits<double>::epsilon()) * pa_cpu[8])
    //         Rynu -= dY;
    //     else
    //         break;

    // std::array<double, 2> conformParams = { 
    //     (Rxnu + Rynu) / 2.0,
    //     std::abs(Rxnu - Rynu) / 2.0
    // };
    // std::sort(conformParams.begin(), conformParams.end());

    // const double c0nu = conformParams[1];
    // const double kappaSignNu = (tauInfty * deltaP > 0.0) ? 1.0 : -1.0;
    // const double kappaNu = kappaSign * conformParams[0] / conformParams[1];

    // std::cout << "kappa num = " << kappaNu << "; kappa an = " << kappa << "\n"
    //           << "c0 num = " << c0nu << "; c0 an = " << c0 << "\n";

    double* Uanr = new double[nX * nY];
    double* Unur = new double[nX * nY];

    for (int i = 0; i < nX; i++)
    {
        for (int j = 0; j < nY; j++)
        {
            const double x = -0.5 * dX * (nX - 1) + dX * i;
            const double y = -0.5 * dY * (nY - 1) + dY * j;
            const double r = sqrt(x * x + y * y);
            const double cosf = x / r;
            const double sinf = y / r;

            if (
                x * x + y * y < Rmin * Rmin ||
                x * x + y * y > Rmax * Rmax
            )
            {
                Uanr[j * nX + i] = 0.0;
                Unur[j * nX + i] = 0.0;
            }
            else
            {
                if (x * x / (Rx * Rx) + y * y / (Ry * Ry) > 0) 
                {
                    const std::complex<double> z = std::complex<double>(x, y);

                    double signx = x > 0.0 ? 1.0 : -1.0;
                    if (abs(x) < std::numeric_limits<double>::epsilon())
                        signx = 1.0;

                    const std::complex<double> zeta = (z + signx * sqrt(z * z + 4.0 * c0 * c0 * kappa)) / 2.0 / c0;
                    const std::complex<double> w = c0 * (zeta - kappa / zeta);
                    const std::complex<double> dw = c0 * (1.0 + kappa / (zeta * zeta));
                    const std::complex<double> wv = c0 * (1.0 / zeta - kappa * zeta);
                    const std::complex<double> Phi = -Y * xi / 2.0 - Y * xi * log(w / zeta / rad); 
                    const std::complex<double> Psi = -Y * xi / zeta * wv / dw;
                    const std::complex<double> phi = - Y * xi * w * (log(w / zeta / rad) + 0.5) - 2.0 * c0 * tauInfty / zeta;
                    const std::complex<double> psi = c0 * Y * xi * (1.0 / zeta + kappa * zeta);
                    const std::complex<double> dphi = Phi * dw;
                    const std::complex<double> dpsi = Psi * dw;
                    const std::complex<double> U = (1.0 / (2.0 * G0) + 3.0 / (G0 + 3.0 * K0)) * phi - w / conj(dw) * conj(dphi) - conj(psi);

                    Uanr[j * nX + i] = real(U) * cosf + imag(U) * sinf;
                }
                else
                {
                    Uanr[j * nX + i] = -0.5 * Y * rad * rad * exp((deltaP - Y) / Y) / (G0 * r);
                }
                
                Unur[j * nX + i] = Ux_cpu[(nX + 1) * j + i] * cosf + Uy_cpu[nX * j + i] * sinf;
            }
        }
    }

    SaveVector(Uanr, nX * nY, "Uanr_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Uanr;

    SaveVector(Unur, nX * nY, "Unur_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Unur;

    double* Sanrr = new double[(nX - 1) * (nY - 1)];
    double* Sanff = new double[(nX - 1) * (nY - 1)];
    double* Sanrf = new double[(nX - 1) * (nY - 1)];

    double* Snurr = new double[(nX - 1) * (nY - 1)];
    double* Snuff = new double[(nX - 1) * (nY - 1)];
    double* Snurf = new double[(nX - 1) * (nY - 1)];

    double* plastZoneAn = new double[(nX - 1) * (nY - 1)];
    double* plastZoneNu = new double[(nX - 1) * (nY - 1)];

    for (int i = 0; i < nX - 1; i++)
    {
        for (int j = 0; j < nY - 1; j++)
        {
            const double x = -0.5 * dX * (nX - 1) + dX * i + 0.5 * dX;
            const double y = -0.5 * dY * (nY - 1) + dY * j + 0.5 * dY;
            const double r = sqrt(x * x + y * y);
            const double cosf = x / r;
            const double sinf = y / r;

            const std::complex<double> z = std::complex<double>(x, y);

            const double Rmin = rad + /*2*/0.0 * std::min(dX, dY);
            const double Rmax = 0.5 * dX * (nX - 1) - dX * 40;
            double signx = x > 0.0 ? 1.0 : -1.0;
            if (abs(x) < std::numeric_limits<double>::epsilon())
                signx = 1.0;

            const std::complex<double> zeta = (z + signx * sqrt(z * z + 4.0 * c0 * c0 * kappa)) / 2.0 / c0;
            const std::complex<double> w = c0 * (zeta - kappa / zeta);
            const std::complex<double> dw = c0 * (1.0 + kappa / (zeta * zeta));
            const std::complex<double> wv = c0 * (1.0 / zeta - kappa * zeta);
            const std::complex<double> phi = -Y * xi / 2.0 - Y * xi * log(w / zeta / rad); 
            const std::complex<double> psi = -Y * xi / zeta * wv / dw;
            const std::complex<double> dphi = - 2.0 * xi * Y * kappa / zeta / ( zeta * zeta - kappa );
            const std::complex<double> F = 2.0 * (conj(w) / dw * dphi + psi) / exp(-2.0 * arg(z) * std::complex<double>(0.0, 1.0));

            plastZoneAn[j * (nX - 1) + i] = 0.0;
            plastZoneNu[j * (nX - 1) + i] = 0.0;

            if (
                x * x + y * y < Rmin * Rmin ||
                x * x + y * y > Rmax * Rmax
            )
            {
                Sanrr[j * (nX - 1) + i] = 0.0;
                Sanff[j * (nX - 1) + i] = 0.0;
                Sanrf[j * (nX - 1) + i] = 0.0;

                Snurr[j * (nX - 1) + i] = 0.0;
                Snuff[j * (nX - 1) + i] = 0.0;
                Snurf[j * (nX - 1) + i] = 0.0;
            }
            else
            {
                const double relR = rad / r;
                const double J2 = 0.25 * (J2_cpu[j * nX + i] + J2_cpu[j * nX + (i + 1)] + J2_cpu[(j + 1) * nX + i] + J2_cpu[(j + 1) * nX + (i + 1)]);

                if (J2 > (1.0 - 2.0 * std::numeric_limits<double>::epsilon()) * pa_cpu[8]) 
                {
                    plastZoneNu[j * (nX - 1) + i] = 1.0;
                }

                if (x * x / (Rx * Rx) + y * y / (Ry * Ry) > 1.0) 
                {
                    // elast
                    // Sanrr[j * (nX - 1) + i] = -deltaP + relR * relR * Y * exp(deltaP / Y - 1);
                    // Sanff[j * (nX - 1) + i] = -deltaP - relR * relR * Y * exp(deltaP / Y - 1);

                    Sanrr[j * (nX - 1) + i] = 2.0 * real(phi) - real(F) / 2.0;
                    Sanff[j * (nX - 1) + i] = 2.0 * real(phi) + real(F) / 2.0;
                    Sanrf[j * (nX - 1) + i] = imag(F) / 2.0;
                }
                else 
                {
                    // plast
                    Sanrr[j * (nX - 1) + i] = -2.0 * Y * log(1.0 / relR);
                    Sanff[j * (nX - 1) + i] = -2.0 * Y * (1.0 + log(1.0 / relR));
                    Sanrf[j * (nX - 1) + i] = 0;

                    plastZoneAn[j * (nX - 1) + i] = 1.0;
                }

                const double Sxx = 0.25 * (
                    -P_cpu[j * nX + i] + tauXX_cpu[j * nX + i] +
                    -P_cpu[j * nX + (i + 1)] + tauXX_cpu[j * nX + (i + 1)] +
                    -P_cpu[(j + 1) * nX + i] + tauXX_cpu[(j + 1) * nX + i] +
                    -P_cpu[(j + 1) * nX + (i + 1)] + tauXX_cpu[(j + 1) * nX + (i + 1)]
                );
                const double Syy = 0.25 * (
                    -P_cpu[j * nX + i] + tauYY_cpu[j * nX + i] +
                    -P_cpu[j * nX + (i + 1)] + tauYY_cpu[j * nX + (i + 1)] +
                    -P_cpu[(j + 1) * nX + i] + tauYY_cpu[(j + 1) * nX + i] +
                    -P_cpu[(j + 1) * nX + (i + 1)] + tauYY_cpu[(j + 1) * nX + (i + 1)]
                );

                const double Sxy = tauXY_cpu[j * (nX - 1) + i];

                Snurr[j * (nX - 1) + i] = Sxx * cosf * cosf + Syy * sinf * sinf + 2 * Sxy * sinf * cosf;
                Snuff[j * (nX - 1) + i] = Sxx * sinf * sinf + Syy * cosf * cosf - 2 * Sxy * sinf * cosf;
                Snurf[j * (nX - 1) + i] = (Syy - Sxx) * sinf * cosf + Sxy * (cosf * cosf - sinf * cosf);
            }
        }
    }

    SaveVector(Sanrr, (nX - 1) * (nY - 1), "Sanrr_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Sanrr;

    SaveVector(Sanff, (nX - 1) * (nY - 1), "Sanff_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Sanff;

    SaveVector(Sanrf, (nX - 1) * (nY - 1), "Sanrf_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Sanrf;

    SaveVector(Snurr, (nX - 1) * (nY - 1), "Snurr_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Snurr;

    SaveVector(Snuff, (nX - 1) * (nY - 1), "Snuff_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Snuff;

    SaveVector(Snurf, (nX - 1) * (nY - 1), "Snurf_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] Snurf;

    SaveVector(plastZoneAn, (nX - 1) * (nY - 1), "plast_an_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] plastZoneAn;

    SaveVector(plastZoneNu, (nX - 1) * (nY - 1), "plast_nu_" + std::to_string(32 * NGRID) + "_.dat");
    delete[] plastZoneNu;
}

EffPlast2D::EffPlast2D() {
    block.x = 32;
    block.y = 32;
    block.z = 1;
    grid.x = NGRID;
    grid.y = NGRID;
    grid.z = 1;

    nX = block.x * grid.x;
    nY = block.y * grid.y;

    gpuErrchk(hipSetDevice(0));
    //gpuErrchk(hipDeviceReset());
    //gpuErrchk(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

    /* PARAMETERS */
    pa_cpu = new double[NPARS];
    gpuErrchk(hipMalloc(&pa_cuda, NPARS * sizeof(double)));
    ReadParams("pa.dat");

    dX = pa_cpu[0];
    dY = pa_cpu[1];
    dT = pa_cpu[2];
    K0 = pa_cpu[3];
    G0 = pa_cpu[4];
    rad = pa_cpu[9];
    Y = pa_cpu[8] / sqrt(2.0);
    N = pa_cpu[10];

    /* SPACE ARRAYS */
    // materials
    K_cpu = new double[nX * nY];
    G_cpu = new double[nX * nY];
    gpuErrchk(hipMalloc(&K_cuda, nX * nY * sizeof(double)));
    gpuErrchk(hipMalloc(&G_cuda, nX * nY * sizeof(double)));
    SetMaterials();

    // stress
    P0_cpu = new double[nX * nY];
    gpuErrchk(hipMalloc(&P0_cuda, nX * nY * sizeof(double)));
    SetInitPressure(pa_cpu[8]);

    SetMatrixZero(&P_cpu, &P_cuda, nX, nY);
    SetMatrixZero(&tauXX_cpu, &tauXX_cuda, nX, nY);
    SetMatrixZero(&tauYY_cpu, &tauYY_cuda, nX, nY);
    SetMatrixZero(&tauXY_cpu, &tauXY_cuda, nX - 1, nY - 1);
    SetMatrixZero(&tauXYav_cpu, &tauXYav_cuda, nX, nY);

    // plasticity
    SetMatrixZero(&J2_cpu, &J2_cuda, nX, nY);
    SetMatrixZero(&J2XY_cpu, &J2XY_cuda, nX - 1, nY - 1);

    // displacement
    SetMatrixZero(&Ux_cpu, &Ux_cuda, nX + 1, nY);
    SetMatrixZero(&Uy_cpu, &Uy_cuda, nX, nY + 1);

    // velocity
    SetMatrixZero(&Vx_cpu, &Vx_cuda, nX + 1, nY);
    SetMatrixZero(&Vy_cpu, &Vy_cuda, nX, nY + 1);

    /* UTILITIES */
    log_file.open("EffPlast2D.log", std::ios_base::app);
    output_step = 10'000;
}

EffPlast2D::~EffPlast2D() {
    // parameters
    delete[] pa_cpu;
    gpuErrchk(hipFree(pa_cuda));

    // materials
    delete[] K_cpu;
    delete[] G_cpu;
    gpuErrchk(hipFree(K_cuda));
    gpuErrchk(hipFree(G_cuda));

    // stress
    delete[] P0_cpu;
    delete[] P_cpu;
    delete[] tauXX_cpu;
    delete[] tauYY_cpu;
    delete[] tauXY_cpu;
    delete[] tauXYav_cpu;
    gpuErrchk(hipFree(P0_cuda));
    gpuErrchk(hipFree(P_cuda));
    gpuErrchk(hipFree(tauXX_cuda));
    gpuErrchk(hipFree(tauYY_cuda));
    gpuErrchk(hipFree(tauXY_cuda));
    gpuErrchk(hipFree(tauXYav_cuda));

    // plasticity
    delete[] J2_cpu;
    delete[] J2XY_cpu;
    gpuErrchk(hipFree(J2_cuda));
    gpuErrchk(hipFree(J2XY_cuda));

    // displacement
    delete[] Ux_cpu;
    delete[] Uy_cpu;
    gpuErrchk(hipFree(Ux_cuda));
    gpuErrchk(hipFree(Uy_cuda));

    // velocity
    delete[] Vx_cpu;
    delete[] Vy_cpu;
    gpuErrchk(hipFree(Vx_cuda));
    gpuErrchk(hipFree(Vy_cuda));

    // log
    log_file.close();
}