#include "hip/hip_runtime.h"
#include "EffPlast2D.h"

__global__ void ComputeDisp(double* Ux, double* Uy, double* Vx, double* Vy, 
                            const double* const P,
                            const double* const tauXX, const double* const tauYY, const double* const tauXY,
                            const double* const pa,
                            const long int nX, const long int nY) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  const double dX = pa[0], dY = pa[1];
  const double dT = pa[2];
  const double rho = pa[5];
  const double dampX = pa[6], dampY = pa[7];

  // motion equation
  if (i > 0 && i < nX && j > 0 && j < nY - 1) {
    Vx[j * (nX + 1) + i] = Vx[j * (nX + 1) + i] * (1.0 - dT * dampX) + (dT / rho) * ( (
                           -P[j * nX + i] + P[j * nX + i - 1] + tauXX[j * nX + i] - tauXX[j * nX + i - 1]
                           ) / dX + (
                           tauXY[j * (nX - 1) + i - 1] - tauXY[(j - 1) * (nX - 1) + i - 1]
                           ) / dY );
  }
  if (i > 0 && i < nX - 1 && j > 0 && j < nY) {
    Vy[j * nX + i] = Vy[j * nX + i] * (1.0 - dT * dampY) + (dT / rho) * ( (
                     -P[j * nX + i] + P[(j - 1) * nX + i] + tauYY[j * nX + i] - tauYY[(j - 1) * nX + i]
                     ) / dY + (
                     tauXY[(j - 1) * (nX - 1) + i] - tauXY[(j - 1) * (nX - 1) + i - 1]
                     ) / dX );
  }

  Ux[j * (nX + 1) + i] = Ux[j * (nX + 1) + i] + Vx[j * (nX + 1) + i] * dT;
  Uy[j * nX + i]       = Uy[j * nX + i]       + Vy[j * nX + i] * dT;
}

__global__ void ComputeStress(const double* const Ux, const double* const Uy,
                              const double* const K, const double* const G,
                              const double* const P0, double* P,
                              double* tauXX, double* tauYY, double* tauXY,
                              const double* const pa,
                              const long int nX, const long int nY) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  const double dX = pa[0], dY = pa[1];
  // const double dT = pa[2];
  const double rad = pa[9];
  const double N = pa[10];

  // constitutive equation - Hooke's law
  P[j * nX + i] = P0[j * nX + i] - K[j * nX + i] * ( 
                  (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY    // divU
                  );

  /*P[j * nX + i] = P[j * nX + i] - G[j * nX + i] * ( // incompressibility
                  (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY    // divU
                  ) * dT / nX;*/

  tauXX[j * nX + i] = 2.0 * G[j * nX + i] * (
                      (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX -    // dUx/dx
                      ( (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY ) / 3.0    // divU / 3.0
                      );
  tauYY[j * nX + i] = 2.0 * G[j * nX + i] * (
                      (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY -    // dUy/dy
                      ( (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY ) / 3.0    // divU / 3.0
                      );

  if (i < nX - 1 && j < nY - 1) {
    tauXY[j * (nX - 1) + i] = 0.25 * (G[j * nX + i] + G[j * nX + i + 1] + G[(j + 1) * nX + i] + G[(j + 1) * nX + i + 1]) * (
                              (Ux[(j + 1) * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i + 1]) / dY + (Uy[(j + 1) * nX + i + 1] - Uy[(j + 1) * nX + i]) / dX    // dUx/dy + dUy/dx
                              );
  }

  for (int k = 0; k < N; k++) {
    for (int l = 0; l < N; l++) {
      if (sqrt((-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) * 
               (-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) + 
               (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l) * 
               (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l)) < rad ) {
        P[j * nX + i] = 0.0;
        tauXX[j * nX + i] = 0.0;
        tauYY[j * nX + i] = 0.0;
      }

      if (i < nX - 1 && j < nY - 1) {
        if (sqrt((-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) * 
                 (-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * k) + 
                 (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l) * 
                 (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * l)) < rad ) {
          tauXY[j * (nX - 1) + i] = 0.0;
        }
      }
    }
  }
}

__global__ void ComputePlasticity(double* tauXX, double* tauYY, double* tauXY,
                                  double* const tauXYav,
                                  double* const J2, double* const J2XY,
                                  const double* const pa,
                                  const long int nX, const long int nY) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  //const double dX = pa[0], dY = pa[1];
  const double coh = pa[8];
  //const double rad = pa[9];

  // tauXY for plasticity
  if (i > 0 && i < nX - 1 && 
      j > 0 && j < nY - 1) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 1) * (nX - 1) + i - 1] + tauXY[(j - 1) * (nX - 1) + i] + tauXY[j * (nX - 1) + i - 1] + tauXY[j * (nX - 1) + i]);
  }
  else if (i == 0 && j > 0 && j < nY - 1) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 1) * (nX - 1) + i] + tauXY[(j - 1) * (nX - 1) + i + 1] + tauXY[j * (nX - 1) + i] + tauXY[j * (nX - 1) + i + 1]);
  }
  else if (i == nX - 1 && j > 0 && j < nY - 1) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 1) * (nX - 1) + i - 2] + tauXY[(j - 1) * (nX - 1) + i - 1] + tauXY[j * (nX - 1) + i - 2] + tauXY[j * (nX - 1) + i - 1]);
  }
  else if (i > 0 && i < nX - 1 && j == 0) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[j * (nX - 1) + i - 1] + tauXY[j * (nX - 1) + i] + tauXY[(j + 1) * (nX - 1) + i - 1] + tauXY[(j + 1) * (nX - 1) + i]);
  }
  else if (i > 0 && i < nX - 1 && j == nY - 1) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 2) * (nX - 1) + i - 1] + tauXY[(j - 2) * (nX - 1) + i] + tauXY[(j - 1) * (nX - 1) + i - 1] + tauXY[(j - 1) * (nX - 1) + i]);
  }
  else if (i == 0 && j == 0) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[j * (nX - 1) + i] + tauXY[j * (nX - 1) + i + 1] + tauXY[(j + 1) * (nX - 1) + i] + tauXY[(j + 1) * (nX - 1) + i + 1]);
  }
  else if (i == 0 && j == nY - 1) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 2) * (nX - 1) + i] + tauXY[(j - 2) * (nX - 1) + i + 1] + tauXY[(j - 1) * (nX - 1) + i] + tauXY[(j - 1) * (nX - 1) + i + 1]);
  }
  else if (i == nX - 1 && j == 0) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[j * (nX - 1) + i - 2] + tauXY[j * (nX - 1) + i - 1] + tauXY[(j + 1) * (nX - 1) + i - 2] + tauXY[(j + 1) * (nX - 1) + i - 1]);
  }
  else if (i == nX - 1 && j == nY - 1) {
    tauXYav[j * nX + i] = 0.25 * (tauXY[(j - 2) * (nX - 1) + i - 2] + tauXY[(j - 2) * (nX - 1) + i - 1] + tauXY[(j - 1) * (nX - 1) + i - 2] + tauXY[(j - 1) * (nX - 1) + i - 1]);
  }

  /*if (sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) < rad ) {
    tauXYav[j * nX + i] = 0.0;
  }*/

  // plasticity
  J2[j * nX + i] = sqrt( tauXX[j * nX + i] * tauXX[j * nX + i] + tauYY[j * nX + i] * tauYY[j * nX + i] + 2.0 * tauXYav[j * nX + i] * tauXYav[j * nX + i] );
  if (i < nX - 1 && j < nY - 1) {
    J2XY[j * (nX - 1) + i] = sqrt(
      0.0625 * (tauXX[j * nX + i] + tauXX[j * nX + i + 1] + tauXX[(j + 1) * nX + i] + tauXX[(j + 1) * nX + i + 1]) * (tauXX[j * nX + i] + tauXX[j * nX + i + 1] + tauXX[(j + 1) * nX + i] + tauXX[(j + 1) * nX + i + 1]) + 
      0.0625 * (tauYY[j * nX + i] + tauYY[j * nX + i + 1] + tauYY[(j + 1) * nX + i] + tauYY[(j + 1) * nX + i + 1]) * (tauYY[j * nX + i] + tauYY[j * nX + i + 1] + tauYY[(j + 1) * nX + i] + tauYY[(j + 1) * nX + i + 1]) + 
      2.0 * tauXY[j * (nX - 1) + i] * tauXY[j * (nX - 1) + i]
    );
  }

  if (J2[j * nX + i] > coh) {
    tauXX[j * nX + i] *= coh / J2[j * nX + i];
    tauYY[j * nX + i] *= coh / J2[j * nX + i];
    tauXYav[j * nX + i] *= coh / J2[j * nX + i];
    J2[j * nX + i] = sqrt(tauXX[j * nX + i] * tauXX[j * nX + i] + tauYY[j * nX + i] * tauYY[j * nX + i] + 2.0 * tauXYav[j * nX + i] * tauXYav[j * nX + i]);
  }

  if (i < nX - 1 && j < nY - 1) {
    if (J2XY[j * (nX - 1) + i] > coh) {
      tauXY[j * (nX - 1) + i] *= coh / J2XY[j * (nX - 1) + i];
    }
  }
}

std::vector< std::array<double, 3> > EffPlast2D::ComputeSigma(const double loadValue, const std::array<double, 3>& loadType) {
  /* INPUT DATA */

  // displacement
  const double dUxdx = loadValue * loadType[0];
  const double dUydy = loadValue * loadType[1];
  const double dUxdy = loadValue * loadType[2];

  //std::cout << "Before loop...\n";

  std::vector< std::array<double, 3> > Sigma(NT);
  for (auto& i : Sigma) {
    i = {0.0, 0.0, 0.0};
  }

  /* ACTION LOOP */
  for (int it = 0; it < NT; it++) {
    hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < nX + 1; i++) {
      for (int j = 0; j < nY; j++) {
        Ux_cpu[j * (nX + 1) + i] += ((-0.5 * dX * nX + dX * i) * dUxdx + (-0.5 * dY * (nY - 1) + dY * j) * dUxdy) / NT;
      }
    }
    hipMemcpy(Ux_cuda, Ux_cpu, (nX + 1) * nY * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < nX; i++) {
      for (int j = 0; j < nY + 1; j++) {
        Uy_cpu[j * nX + i] += (-0.5 * dY * nY + dY * j) * dUydy / NT;
      }
    }
    hipMemcpy(Uy_cuda, Uy_cpu, nX * (nY + 1) * sizeof(double), hipMemcpyHostToDevice);

    double error = 0.0;

    /* ITERATION LOOP */
    for (int iter = 0; iter < NITER; iter++) {
      ComputeStress<<<grid, block>>>(Ux_cuda, Uy_cuda, K_cuda, G_cuda, P0_cuda, P_cuda, tauXX_cuda, tauYY_cuda, tauXY_cuda, /*tauXYav_cuda, J2_cuda, J2XY_cuda,*/ pa_cuda, nX, nY);
      hipDeviceSynchronize();    // wait for compute device to finish
      ComputePlasticity<<<grid, block>>>(tauXX_cuda, tauYY_cuda, tauXY_cuda, tauXYav_cuda, J2_cuda, J2XY_cuda, pa_cuda, nX, nY);
      hipDeviceSynchronize();    // wait for compute device to finish
      //std::cout << "After computing sigma...\n";
      ComputeDisp<<<grid, block>>>(Ux_cuda, Uy_cuda, Vx_cuda, Vy_cuda, P_cuda, tauXX_cuda, tauYY_cuda, tauXY_cuda, pa_cuda, nX, nY);
      hipDeviceSynchronize();    // wait for compute device to finish

      if ((iter + 1) % output_step == 0) {
        hipMemcpy(Vx_cpu, Vx_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(Vy_cpu, Vy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost);
        error = (FindMaxAbs(Vx_cpu, (nX + 1) * nY) / (dX * (nX - 1)) + FindMaxAbs(Vy_cpu, nX * (nY + 1)) / (dY * (nY - 1))) * dT /
          (std::abs(loadValue) * std::max( std::max(std::abs(loadType[0]), std::abs(loadType[1])), std::abs(loadType[2]) ));
        std::cout << "Iteration " << iter + 1 << ": Error is " << error << '\n';
        // log_file << "Iteration " << iter + 1 << ": Error is " << error << '\n';
        if (error < EITER) {
          std::cout << "Number of iterations is " << iter + 1 << '\n';
          log_file << "Number of iterations is " << iter + 1 << '\n';
          break;
        }
        else if (iter == NITER - 1) {
          std::cout << "WARNING: Maximum number of iterations reached!\nError is " << error << '\n';
          log_file << "WARNING: Maximum number of iterations reached!\nError is " << error << '\n';
        }
        // std::cout << "Vx on step " << it << " is " << Vx_cpu[nY/2 * (nX + 1) + nX/2] << std::endl;
        // log_file << "Vx on step " << it << " is " << Vx_cpu[nY/2 * (nX + 1) + nX/2] << std::endl;
      }
    }
    /* AVERAGING */
    hipMemcpy(P_cpu, P_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(tauXX_cpu, tauXX_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(tauYY_cpu, tauYY_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(tauXY_cpu, tauXY_cuda, (nX - 1) * (nY - 1) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(J2_cpu, J2_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost);

    /*for (int i = 0; i < nX; i++) {
      for (int j = 0; j < nY; j++) {
        Sigma[it][0] += tauXX_cpu[j * nX + i] - P_cpu[j * nX + i];
        Sigma[it][1] += tauYY_cpu[j * nX + i] - P_cpu[j * nX + i];
      }
    }
    Sigma[it][0] /= nX * nY;
    Sigma[it][1] /= nX * nY;

    for (int i = 0; i < nX - 1; i++) {
      for (int j = 0; j < nY - 1; j++) {
        Sigma[it][2] += tauXY_cpu[j * (nX - 1) + i];
      }
    }
    Sigma[it][2] /= (nX - 1) * (nY - 1);*/

    // -P_eff
    for (int i = 0; i < nX; i++) {
      for (int j = 0; j < nY; j++) {
        if ( sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) >= rad ) {
          Sigma[it][0] += - P_cpu[j * nX + i];
        }
        else {
          // std::cout << "In the hole!\n";
          // log_file << "In the hole!\n";
        }
      }
    }
    Sigma[it][0] /= nX * nY;

    // Tau_eff
    for (int i = 0; i < nX; i++) {
      for (int j = 0; j < nY; j++) {
        if ( sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) >= rad ) {
          Sigma[it][1] += tauXX_cpu[j * nX + i];
          Sigma[it][2] += tauYY_cpu[j * nX + i];
        }
      }
    }
    Sigma[it][1] /= nX * nY;
    Sigma[it][2] /= nX * nY;

    // std::cout << Sigma[it][0] / loadValue << '\t' << Sigma[it][1] / loadValue << '\t' << Sigma[it][2] / loadValue << '\n';
    // log_file << Sigma[it][0] / loadValue << '\t' << Sigma[it][1] / loadValue << '\t' << Sigma[it][2] / loadValue << '\n';

    /* ANALYTIC SOLUTION FOR EFFECTIVE PROPERTIES */
    const double deltaP = GetDeltaP_honest();
    //const double deltaP = GetDeltaP_approx(loadValue * loadType[0], loadValue * loadType[1]);
    const double tauInfty_approx = GetTauInfty_approx(loadValue * loadType[0], loadValue * loadType[1]);

    int holeX = static_cast<int>((nX + 1) * 2 * rad / nX / dX);    // approx X-axis index of hole boundary
    std::vector<double> dispX((nX + 1) / 2);
    for (int i = (nX + 1) / 2 - holeX - 1; i < (nX + 1) / 2; i++) {
      dispX[i] = Ux_cpu[(nY / 2) * (nX + 1) + i];
    }

    int holeY = static_cast<int>((nY + 1) * 2 * rad / nY / dY);    // approx Y-axis index of hole boundary
    std::vector<double> dispY((nY + 1) / 2);
    for (int j = (nY + 1) / 2 - holeY - 1; j < (nY + 1) / 2; j++) {
      dispY[j] = Uy_cpu[j * nX + nX / 2];
    }

    /*std::vector<double> dispXwrong((nY + 1) / 2);
    for (int j = nY / 2 - holeY - 2; j < nY / 2; j++) {
      dispXwrong[j] = Ux_cpu[j * nX + nX / 2];
    }*/

    /*const double dR = FindMaxAbs(Ux_cpu, (nX + 1) * nY);
    std::cout << "dR = " << dR << '\n';
    log_file << "dR = " << dR << '\n';*/
    const double dRx = -FindMaxAbs(dispX);
    std::cout << "dRx = " << dRx << '\n';
    log_file << "dRx = " << dRx << '\n';
    const double dRy = -FindMaxAbs(dispY);
    std::cout << "dRy = " << dRy << '\n';
    log_file << "dRy = " << dRy << '\n';
    /*const double dRxWrong = -FindMaxAbs(dispXwrong);
    std::cout << "dRxWrong = " << dRxWrong << '\n';*/
    const double Phi0 = 3.1415926 * rad * rad / (dX * (nX - 1) * dY * (nY - 1));
    const double Phi = 3.1415926 * (rad + dRx) * (rad + dRy) / (dX * (nX - 1) * dY * (nY - 1) * (1 + loadValue * loadType[0]) * (1 + loadValue * loadType[1]));
    const double dPhi = 3.1415926 * ( std::abs((rad + dRx) * (rad + dRy) - rad * rad) ) / (dX * (nX - 1) * dY * (nY - 1));
    // std::cout << "dPhi = " << dPhi << '\n';
    // log_file << "dPhi = " << dPhi << '\n';

    const double KeffPhi = deltaP / dPhi;
    //const double KeffPhi = deltaP_honest / dPhi;
    
    //std::cout << "deltaP_honest = " << deltaP_honest << '\n';
    //log_file << "deltaP_honest = " << deltaP_honest << '\n';
    std::cout << "deltaP / Y = " << deltaP / Y << '\n';
    log_file << "deltaP / Y = " << deltaP / Y << '\n';
    std::cout << "tauInfty / Y = " << tauInfty_approx / Y << '\n';
    log_file << "tauInfty / Y = " << tauInfty_approx / Y << '\n';
    std::cout << "KeffPhi = " << KeffPhi << '\n';
    log_file << "KeffPhi = " << KeffPhi << '\n';

    const double phi = 3.1415926 * rad * rad / (dX * (nX - 1) * dY * (nY - 1));
    const double KexactElast = G0 / phi;
    const double KexactPlast = G0 / (phi - dPhi) / exp(std::abs(deltaP) / Y - 1.0) / // phi or phi - dPhi ?
      (1.0 + 5.0 * tauInfty_approx * tauInfty_approx / Y / Y);
    //const double KexactPlast = G0 / phi / exp(std::abs(deltaP_honest) / pa_cpu[8] - 1.0);
    std::cout << "KexactElast = " << KexactElast << '\n';
    log_file << "KexactElast = " << KexactElast << '\n';
    std::cout << "KexactPlast = " << KexactPlast << '\n';
    log_file << "KexactPlast = " << KexactPlast << '\n';

    SaveAnStatic1D(deltaP);
  }

  /* ANALYTIC 2D SOLUTION FOR STATICS */

  /* OUTPUT DATA WRITING */
  SaveMatrix(P_cpu, P_cuda, nX, nY, "Pc_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(tauXX_cpu, tauXX_cuda, nX, nY, "tauXXc_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(tauYY_cpu, tauYY_cuda, nX, nY, "tauYYc_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(tauXY_cpu, tauXY_cuda, nX - 1, nY - 1, "tauXYc_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(tauXYav_cpu, tauXYav_cuda, nX, nY, "tauXYavc_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(J2_cpu, J2_cuda, nX, nY, "J2c_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(Ux_cpu, Ux_cuda, nX + 1, nY, "Uxc_" + std::to_string(32 * NGRID) + "_.dat");
  SaveMatrix(Uy_cpu, Uy_cuda, nX, nY + 1, "Uyc_" + std::to_string(32 * NGRID) + "_.dat");

  hipDeviceReset();
  return Sigma;
}

void EffPlast2D::ReadParams(const std::string& filename) {
  FILE* pa_fil = fopen(filename.c_str(), "rb");
  if (!pa_fil) {
    std::cerr << "Error! Cannot open file pa.dat!\n";
    exit(1);
  }
  fread(pa_cpu, sizeof(double), NPARS, pa_fil);
  fclose(pa_fil);
  hipMemcpy(pa_cuda, pa_cpu, NPARS * sizeof(double), hipMemcpyHostToDevice);
}

void EffPlast2D::SetMaterials() {
  //constexpr double K0 = 10.0;
  //constexpr double G0 = 0.01;

  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      K_cpu[j * nX + i] = K0;
      G_cpu[j * nX + i] = G0;
      double x = -0.5 * dX * (nX - 1) + dX * i;
      double y = -0.5 * dY * (nY - 1) + dY * j;
      double Lx = dX * (nX - 1);
      double Ly = dY * (nY - 1);
      for (int k = 0; k < N; k++) {
        for (int l = 0; l < N; l++) {
          if ( sqrt((x - 0.5 * Lx * (1.0 - 1.0/N) + (Lx/N)*k) * (x - 0.5 * Lx * (1.0 - 1.0/N) + (Lx/N)*k) + 
                    (y - 0.5 * Ly * (1.0 - 1.0/N) + (Ly/N)*l) * (y - 0.5 * Ly * (1.0 - 1.0/N) + (Ly/N)*l)) < rad ) {
            K_cpu[j * nX + i] = 0.01 * K0;
            G_cpu[j * nX + i] = 0.01 * G0;
          }
        }
      }
    }
  }

  hipMemcpy(K_cuda, K_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(G_cuda, G_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice);
}

void EffPlast2D::SetInitPressure(const double coh) {
  const double P0 = 0.0; //1.0 * coh;

  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      P0_cpu[j * nX + i] = 0.0;
      if ( sqrt((-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j)) < rad ) {
        P0_cpu[j * nX + i] = P0;
      }
    }
  }

  hipMemcpy(P0_cuda, P0_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice);
}

void EffPlast2D::SetMatrixZero(double** A_cpu, double** A_cuda, const int m, const int n) {
  *A_cpu = (double*)malloc(m * n * sizeof(double));
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      (*A_cpu)[j * m + i] = 0.0;
    }
  }
  hipMalloc(A_cuda, m * n * sizeof(double));
  hipMemcpy(*A_cuda, *A_cpu, m * n * sizeof(double), hipMemcpyHostToDevice);
}

void EffPlast2D::SaveMatrix(double* const A_cpu, const double* const A_cuda, const int m, const int n, const std::string& filename) {
  hipMemcpy(A_cpu, A_cuda, m * n * sizeof(double), hipMemcpyDeviceToHost);
  FILE* A_filw = fopen(filename.c_str(), "wb");
  fwrite(A_cpu, sizeof(double), m * n, A_filw);
  fclose(A_filw);
}

void EffPlast2D::SaveVector(double* const arr, const int size, const std::string& filename) {
  FILE* arr_filw = fopen(filename.c_str(), "wb");
  fwrite(arr, sizeof(double), size, arr_filw);
  fclose(arr_filw);
}

double EffPlast2D::FindMaxAbs(const double* const arr, const int size) {
  double max_el = 0.0;
  for (int i = 0; i < size; i++) {
    if (std::abs(arr[i]) > max_el) {
      max_el = std::abs(arr[i]);
    }
  }
  return max_el;
}

double EffPlast2D::FindMaxAbs(const std::vector<double>& vec) {
  double max_el = 0.0;
  for (auto i : vec) {
    if (std::abs(i) > max_el) {
      max_el = i;
    }
  }
  return max_el;
}

double EffPlast2D::GetDeltaP_honest() {
  double deltaP = 0.0, deltaPx = 0.0, deltaPy = 0.0;

  for (int i = 1; i < nX - 1; i++) {
    deltaPx += tauXX_cpu[0 * nX + i] - P_cpu[0 * nX + i];
    deltaPx += tauYY_cpu[0 * nX + i] - P_cpu[0 * nX + i];
    deltaPx += tauXX_cpu[(nY - 1) * nX + i] - P_cpu[(nY - 1) * nX + i];
    deltaPx += tauYY_cpu[(nY - 1) * nX + i] - P_cpu[(nY - 1) * nX + i];
  }
  deltaPx /= (nX - 2);

  for (int j = 1; j < nY - 1; j++) {
    deltaPy += tauXX_cpu[j * nX + 0] - P_cpu[j * nX + 0];
    deltaPy += tauYY_cpu[j * nX + 0] - P_cpu[j * nX + 0];
    deltaPy += tauXX_cpu[j * nX + nY - 1] - P_cpu[j * nX + nY - 1];
    deltaPy += tauYY_cpu[j * nX + nY - 1] - P_cpu[j * nX + nY - 1];
  }
  deltaPy /= (nY - 2);

  deltaP = -0.125 * (deltaPx + deltaPy);
  return deltaP;
}

double EffPlast2D::GetDeltaP_approx(const double Exx, const double Eyy) {
  double deltaP = 0.0;

  if (Exx < Eyy ) {
    deltaP += tauXX_cpu[(nY/2) * nX + 0] - P_cpu[(nY/2) * nX + 0];
    deltaP += tauYY_cpu[(nY/2) * nX + 0] - P_cpu[(nY/2) * nX + 0];
    deltaP += tauXX_cpu[(nY/2) * nX + nX - 1] - P_cpu[(nY/2) * nX + nX - 1];
    deltaP += tauYY_cpu[(nY/2) * nX + nX - 1] - P_cpu[(nY/2) * nX + nX - 1];
  }
  else {
    deltaP += tauXX_cpu[0 * nX + nX/2] - P_cpu[0 * nX + nX/2];
    deltaP += tauYY_cpu[0 * nX + nX/2] - P_cpu[0 * nX + nX/2];
    deltaP += tauXX_cpu[(nY - 1) * nX + nX/2] - P_cpu[(nY - 1) * nX + nX/2];
    deltaP += tauYY_cpu[(nY - 1) * nX + nX/2] - P_cpu[(nY - 1) * nX + nX/2];
  }

  deltaP *= -0.25;
  return deltaP;
}

double EffPlast2D::GetTauInfty_approx(const double Exx, const double Eyy) {
  double tauInfty = 0.0;

  if (Exx < Eyy ) {
    tauInfty += tauYY_cpu[(nY/2) * nX + 0] - tauXX_cpu[(nY/2) * nX + 0];
    tauInfty += tauYY_cpu[(nY/2) * nX + nX - 1] - tauXX_cpu[(nY/2) * nX + nX - 1];
  }
  else {
    tauInfty += tauYY_cpu[0 * nX + nX/2] - tauXX_cpu[0 * nX + nX/2];
    tauInfty += tauYY_cpu[(nY - 1) * nX + nX/2] - tauXX_cpu[(nY - 1) * nX + nX/2];
  }

  tauInfty *= 0.25;
  return tauInfty;
}

void EffPlast2D::SaveAnStatic1D(const double deltaP) {
  /* ANALYTIC 1D SOLUTION FOR STATICS */
  double* xxx = new double[nX];
  for (int i = 0; i < nX; i++) {
    xxx[i] = -0.5 * dX * (nX - 1) + dX * i;
  }
  SaveVector(xxx, nX, "xxx_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] xxx;

  double* Uanr = new double[nX];
  double coef = - 0.5 * Y * rad * rad * exp((deltaP - Y) / Y) / G0;
  for (int i = 0; i < nX; i++) {
    if (std::abs(-0.5 * dX * (nX - 1) + dX * i) < rad) {
      Uanr[i] = 0.0;
    }
    else {
      //Uanr[i] = -0.5 * deltaP * (xxx[i] / (K0 + G0/3.0) + rad * rad / (G0 * xxx[i]));
      Uanr[i] = coef / xxx[i];
    }
  }
  SaveVector(Uanr, nX, "Uanr_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] Uanr;

  double* Unur = new double[nX];
  for (int i = 0; i < nX; i++) {
    Unur[i] = Ux_cpu[nY * (nX + 1) / 2 + i];
  }
  SaveVector(Unur, nX, "Unur_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] Unur;


  double* Sanrr = new double[(nX - 1) * (nY - 1)];
  double* Sanff = new double[(nX - 1) * (nY - 1)];

  double* Snurr = new double[(nX - 1) * (nY - 1)];
  double* Snuff = new double[(nX - 1) * (nY - 1)];

  double* plastZone = new double[(nX - 1) * (nY - 1)];

  for (int i = 0; i < nX - 1; i++)
  {
    for (int j = 0; j < nY - 1; j++)
    {
      const double x = -0.5 * dX * (nX - 1) + dX * i + 0.5 * dX; 
      const double y = -0.5 * dY * (nY - 1) + dY * j + 0.5 * dY;
      const double r = sqrt(x * x + y * y);
      const double cosf = x / r;
      const double sinf = y / r;

      plastZone[j * (nX - 1) + i] = 0.0;

      const double dr = 20.0 * std::min(dX, dY);
      const double R_max = 0.4 * dX * (nX - 3);

      if (
      	x * x + y * y < (rad + dr) * (rad + dr) ||
      	x * x + y * y > R_max * R_max
      )
      {
        Sanrr[j * (nX - 1) + i] = 0.0;
        Sanff[j * (nX - 1) + i] = 0.0;

        Snurr[j * (nX - 1) + i] = 0.0;
        Snuff[j * (nX - 1) + i] = 0.0;
      }
      else
      {
        const double relR = rad / r;
        const double J2 = 0.25 * (J2_cpu[j * nX + i] + J2_cpu[j * nX + (i + 1)] + J2_cpu[(j + 1) * nX + i] + J2_cpu[(j + 1) * nX + (i + 1)]);

        if (J2 <= (1.0 - 2.0 * std::numeric_limits<double>::epsilon()) * pa_cpu[8]) {
          Sanrr[j * (nX - 1) + i] = -deltaP + relR * relR * Y * exp(deltaP / Y - 1);
          Sanff[j * (nX - 1) + i] = -deltaP - relR * relR * Y * exp(deltaP / Y - 1);

        }
        else {
          Sanrr[j * (nX - 1) + i] = -2.0 * Y * log(1.0 / relR);
          Sanff[j * (nX - 1) + i] = -2.0 * Y * (1.0 + log(1.0 / relR));

          plastZone[j * (nX - 1) + i] = 1.0;
        }

        const double Sxx = 0.25 * (
          -P_cpu[j * nX + i] + tauXX_cpu[j * nX + i] + 
          -P_cpu[j * nX + (i + 1)] + tauXX_cpu[j * nX + (i + 1)] + 
          -P_cpu[(j + 1) * nX + i] + tauXX_cpu[(j + 1) * nX + i] + 
          -P_cpu[(j + 1) * nX + (i + 1)] + tauXX_cpu[(j + 1) * nX + (i + 1)]
        );
        const double Syy = 0.25 * (
          -P_cpu[j * nX + i] + tauYY_cpu[j * nX + i] + 
          -P_cpu[j * nX + (i + 1)] + tauYY_cpu[j * nX + (i + 1)] + 
          -P_cpu[(j + 1) * nX + i] + tauYY_cpu[(j + 1) * nX + i] + 
          -P_cpu[(j + 1) * nX + (i + 1)] + tauYY_cpu[(j + 1) * nX + (i + 1)]
        );

        const double Sxy = tauXY_cpu[j * (nX - 1) + i];

        Snurr[j * (nX - 1) + i] = Sxx * cosf * cosf + Syy * sinf * sinf + 2 * Sxy * sinf * cosf;
        Snuff[j * (nX - 1) + i] = Sxx * sinf * sinf + Syy * cosf * cosf - 2 * Sxy * sinf * cosf;
      }
    }
  }

  SaveVector(Sanrr, (nX - 1) * (nY - 1), "Sanrr_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] Sanrr;

  SaveVector(Sanff, (nX - 1) * (nY - 1), "Sanff_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] Sanff;

  SaveVector(Snurr, (nX - 1) * (nY - 1), "Snurr_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] Snurr;

  SaveVector(Snuff, (nX - 1) * (nY - 1), "Snuff_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] Snuff;

  SaveVector(plastZone, (nX - 1) * (nY - 1), "plast_" + std::to_string(32 * NGRID) + "_.dat");
  delete[] plastZone;

  // double* Sanrr = new double[nX];
  // for (int i = 0; i < nX; i++) {
  //   if (std::abs(-0.5 * dX * (nX - 1) + dX * i) <= rad) {
  //     Sanrr[i] = 0.0;
  //   }
  //   else {
  //     double relR = rad / (-0.5 * dX * (nX - 1) + dX * i);
  //     //Sanrr[i] = -deltaP + deltaP * relR * relR - tauInfty_approx * (1.0 - 4.0 * relR * relR + 3.0 * pow(relR, 4.0));
  //     if (J2_cpu[nY * nX / 2 + i] <= (1.0 - 2.0 * std::numeric_limits<double>::epsilon()) * pa_cpu[8]) {
  //       Sanrr[i] = -deltaP + relR * relR * Y * exp(deltaP / Y - 1);
  //     }
  //     else {
  //       Sanrr[i] = -2.0 * Y * log(1.0 / relR);
  //     }
  //   }
  // }
  // SaveVector(Sanrr, nX * nY, "Sanrr_" + std::to_string(32 * NGRID) + "_.dat");
  // delete[] Sanrr;
  //
  // double* Sanff = new double[nX];
  // for (int i = 0; i < nX; i++) {
  //   if (std::abs(-0.5 * dX * (nX - 1) + dX * i) <= rad) {
  //     Sanff[i] = 0.0;
  //   }
  //   else {
  //     double relR = rad / (-0.5 * dX * (nX - 1) + dX * i);
  //     //Sanff[i] = -deltaP - deltaP * relR * relR + tauInfty_approx * (1.0 + 3.0 * pow(relR, 4.0));
  //     if (J2_cpu[nY * nX / 2 + i] <= (1.0 - 2.0 * std::numeric_limits<double>::epsilon()) * pa_cpu[8]) {
  //       Sanff[i] = -deltaP - relR * relR * Y * exp(deltaP / Y - 1);
  //     }
  //     else {
  //       Sanff[i] = -2.0 * Y * (1.0 + log(1.0 / relR));
  //     }
  //   }
  // }
  // SaveVector(Sanff, nX, "Sanff_" + std::to_string(32 * NGRID) + "_.dat");
  // delete[] Sanff;
  //
  // double* Snurr = new double[nX];
  // for (int i = 0; i < nX; i++) {
  //   Snurr[i] = -P_cpu[nY * nX / 2 + i] + tauXX_cpu[nY * nX / 2 + i];
  //   // std::cout << Snurr[i] << '\n';
  // }
  // SaveVector(Snurr, nX, "Snurr_" + std::to_string(32 * NGRID) + "_.dat");
  // delete[] Snurr;

  // double* Snuff = new double[nX];
  // for (int i = 0; i < nX; i++) {
  //   Snuff[i] = -P_cpu[nY * nX / 2 + i] + tauYY_cpu[nY * nX / 2 + i];
  // }
  // SaveVector(Snuff, nX, "Snuff_" + std::to_string(32 * NGRID) + "_.dat");
  // delete[] Snuff;
}

EffPlast2D::EffPlast2D() {
  block.x = 32; 
  block.y = 32; 
  grid.x = NGRID;
  grid.y = NGRID;

  nX = block.x * grid.x;
  nY = block.y * grid.y;

  hipSetDevice(0);
  hipDeviceReset();
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  /* PARAMETERS */
  pa_cpu = (double*)malloc(NPARS * sizeof(double));
  hipMalloc((void**)&pa_cuda, NPARS * sizeof(double));
  ReadParams("pa.dat");

  dX = pa_cpu[0];
  dY = pa_cpu[1];
  dT = pa_cpu[2];
  K0 = pa_cpu[3];
  G0 = pa_cpu[4];
  rad = pa_cpu[9];
  Y = pa_cpu[8] / sqrt(2.0);
  N = pa_cpu[10];

  /* SPACE ARRAYS */
  // materials
  K_cpu = (double*)malloc(nX * nY * sizeof(double));
  G_cpu = (double*)malloc(nX * nY * sizeof(double));
  hipMalloc(&K_cuda, nX * nY * sizeof(double));
  hipMalloc(&G_cuda, nX * nY * sizeof(double));
  SetMaterials();

  // stress
  P0_cpu = (double*)malloc(nX * nY * sizeof(double));
  hipMalloc(&P0_cuda, nX * nY * sizeof(double));
  SetInitPressure(pa_cpu[8]);

  SetMatrixZero(&P_cpu, &P_cuda, nX, nY);
  SetMatrixZero(&tauXX_cpu, &tauXX_cuda, nX, nY);
  SetMatrixZero(&tauYY_cpu, &tauYY_cuda, nX, nY);
  SetMatrixZero(&tauXY_cpu, &tauXY_cuda, nX - 1, nY - 1);
  SetMatrixZero(&tauXYav_cpu, &tauXYav_cuda, nX, nY);

  // plasticity
  SetMatrixZero(&J2_cpu, &J2_cuda, nX, nY);
  SetMatrixZero(&J2XY_cpu, &J2XY_cuda, nX - 1, nY - 1);

  // displacement
  SetMatrixZero(&Ux_cpu, &Ux_cuda, nX + 1, nY);
  SetMatrixZero(&Uy_cpu, &Uy_cuda, nX, nY + 1);

  // velocity
  SetMatrixZero(&Vx_cpu, &Vx_cuda, nX + 1, nY);
  SetMatrixZero(&Vy_cpu, &Vy_cuda, nX, nY + 1);

  /* UTILITIES */
  log_file.open("EffPlast2D.log");
  output_step = 1000;
}

EffPlast2D::~EffPlast2D() {
  // parameters
  free(pa_cpu);
  hipFree(pa_cuda);

  // materials
  free(K_cpu);
  free(G_cpu);
  hipFree(K_cuda);
  hipFree(G_cuda);

  // stress
  free(P0_cpu);
  free(P_cpu);
  free(tauXX_cpu);
  free(tauYY_cpu);
  free(tauXY_cpu);
  free(tauXYav_cpu);
  hipFree(P0_cuda);
  hipFree(P_cuda);
  hipFree(tauXX_cuda);
  hipFree(tauYY_cuda);
  hipFree(tauXY_cuda);
  hipFree(tauXYav_cuda);

  // plasticity
  free(J2_cpu);
  free(J2XY_cpu);
  hipFree(J2_cuda);
  hipFree(J2XY_cuda);

  // displacement
  free(Ux_cpu);
  free(Uy_cpu);
  hipFree(Ux_cuda);
  hipFree(Uy_cuda);

  // velocity
  free(Vx_cpu);
  free(Vy_cpu);
  hipFree(Vx_cuda);
  hipFree(Vy_cuda);

  // log
  log_file.close();
}