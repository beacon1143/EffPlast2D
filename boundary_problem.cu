// Wave 2D 
#include "stdio.h"
#include "stdlib.h"
#include <iostream>
#include <fstream>
#include "hip/hip_runtime.h"

#define NGRID  1
#define NPARS  6
#define NT  10

__global__ void ComputeV(double* Vx, double* Vy, 
                         double* P,
                         double* tauXX, double* tauYY, double* tauXY,
                         double* pa,
                         const long int nX, const long int nY) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  const double dX = pa[0], dY = pa[1];
  const double dT = pa[2];
  const double /*K = pa[3], G = pa[4],*/ rho = pa[5];

  // motion equation
  if (i > 0 && i < nX && j > 0 && j < nY - 1) {
    Vx[j * (nX + 1) + i] = Vx[j * (nX + 1) + i] + (dT / rho) * ( (
                           -P[j * nX + i] + P[j * nX + i - 1] + tauXX[j * nX + i] - tauXX[j * nX + i - 1]
                           ) / dX + (
                           tauXY[j * (nX - 1) + i - 1] - tauXY[(j - 1) * (nX - 1) + i - 1]
                           ) / dY );
  }
  if (i > 0 && i < nX - 1 && j > 0 && j < nY) {
    Vy[j * nX + i] = Vy[j * nX + i] + (dT / rho) * ( (
                     -P[j * nX + i] + P[(j - 1) * nX + i] + tauYY[j * nX + i] - tauYY[(j - 1) * nX + i]
                     ) / dY + (
                     tauXY[(j - 1) * (nX - 1) + i] - tauXY[(j - 1) * (nX - 1) + i - 1]
                     ) / dX );
  }
}

__global__ void ComputeSigma(double* Vx, double* Vy, 
                             double* P,
                             double* tauXX, double* tauYY, double* tauXY,
                             double* pa,
                             const long int nX, const long int nY) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  const double dX = pa[0], dY = pa[1];
  const double dT = pa[2];
  const double K = pa[3], G = pa[4]/*, rho = pa[5]*/;

  // constitutive equation - Hooke's law
  P[j * nX + i] = P[j * nX + i] + (-K * ( 
                  (Vx[j * (nX + 1) + i + 1] - Vx[j * (nX + 1) + i]) / dX + (Vy[(j + 1) * nX + i] - Vy[j * nX + i]) / dY    // divV
                  ) ) * dT;

  tauXX[j * nX + i] = tauXX[j * nX + i] + 2.0 * G * (
                      (Vx[j * (nX + 1) + i + 1] - Vx[j * (nX + 1) + i]) / dX -    // dVdx
                      ( (Vx[j * (nX + 1) + i + 1] - Vx[j * (nX + 1) + i]) / dX + (Vy[(j + 1) * nX + i] - Vy[j * nX + i]) / dY ) / 3.0    // divV / 3.0
                      ) * dT;
  tauYY[j * nX + i] = tauYY[j * nX + i] + 2.0 * G * (
                      (Vy[(j + 1) * nX + i] - Vy[j * nX + i]) / dY -    // dVdy
                      ( (Vx[j * (nX + 1) + i + 1] - Vx[j * (nX + 1) + i]) / dX + (Vy[(j + 1) * nX + i] - Vy[j * nX + i]) / dY ) / 3.0    // divV / 3.0
                      ) * dT;

  if (i < nX - 1 && j < nY - 1) {
    tauXY[j * (nX - 1) + i] = tauXY[j * (nX - 1) + i] + G * (
                              (Vx[(j + 1) * (nX + 1) + i + 1] - Vx[j * (nX + 1) + i + 1]) / dY + (Vy[(j + 1) * nX + i + 1] - Vy[(j + 1) * nX + i]) / dX
                              ) * dT;
  }
}

int main() {
  dim3 grid, block;
  block.x = 32; 
  block.y = 32; 
  grid.x = NGRID;
  grid.y = NGRID;

  const long int nX = block.x * grid.x;
  const long int nY = block.y * grid.y;

  hipSetDevice(0);
  hipDeviceReset();
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  /* INPUT DATA READING */
  // parameters
  double* pa_cuda;
  double* pa_cpu = (double*)malloc(NPARS * sizeof(double));
  //std::ifstream pa_fil("pa.dat", std::ifstream::in | std::ifstream::binary);
  FILE* pa_fil = fopen("pa.dat", "rb");
  if (!pa_fil) {
    std::cerr << "Error! Cannot open file pa.dat!\n";
    return 1;
  }
  //pa_fil.read(pa_cpu, NPARS * sizeof(double));
  fread(pa_cpu, sizeof(double), NPARS, pa_fil);
  //pa_fil.close();
  fclose(pa_fil);
  hipMalloc((void**)&pa_cuda, NPARS * sizeof(double));
  hipMemcpy(pa_cuda, pa_cpu, NPARS * sizeof(double), hipMemcpyHostToDevice);

  // stress
  double* P_cuda;
  double* P_cpu = (double*)malloc(nX * nY * sizeof(double));
  FILE* P_fil = fopen("P.dat", "rb");
  if (!P_fil) {
    std::cerr << "Error! Cannot open file P.dat!\n";
    return 1;
  }
  fread(P_cpu, sizeof(double), nX * nY, P_fil);
  fclose(P_fil);
  hipMalloc(&P_cuda, nX * nY * sizeof(double));
  hipMemcpy(P_cuda, P_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice);

  double* tauXX_cuda;
  double* tauXX_cpu = (double*)malloc(nX * nY * sizeof(double));
  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      tauXX_cpu[j * nX + i] = 0.0;
    }
  }
  hipMalloc(&tauXX_cuda, nX * nY * sizeof(double));
  hipMemcpy(tauXX_cuda, tauXX_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice);

  double* tauYY_cuda;
  double* tauYY_cpu = (double*)malloc(nX * nY * sizeof(double));
  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      tauYY_cpu[j * nX + i] = 0.0;
    }
  }
  hipMalloc(&tauYY_cuda, nX * nY * sizeof(double));
  hipMemcpy(tauYY_cuda, tauYY_cpu, nX * nY * sizeof(double), hipMemcpyHostToDevice);

  double* tauXY_cuda;
  double* tauXY_cpu = (double*)malloc((nX - 1) * (nY - 1) * sizeof(double));
  for (int i = 0; i < nX - 1; i++) {
    for (int j = 0; j < nY - 1; j++) {
      tauXY_cpu[j * (nX - 1) + i] = 0.0;
    }
  }
  hipMalloc(&tauXY_cuda, (nX - 1) * (nY - 1) * sizeof(double));
  hipMemcpy(tauXY_cuda, tauXY_cpu, (nX - 1) * (nY - 1) * sizeof(double), hipMemcpyHostToDevice);

  // velocity
  double* Vx_cuda;
  double* Vx_cpu = (double*)malloc((nX+1) * nY * sizeof(double));
  for (int i = 0; i < nX + 1; i++) {
    for (int j = 0; j < nY; j++) {
      Vx_cpu[j * (nX + 1) + i] = 0.0;
    }
  }
  hipMalloc(&Vx_cuda, (nX + 1) * nY * sizeof(double));
  hipMemcpy(Vx_cuda, Vx_cpu, (nX + 1) * nY * sizeof(double), hipMemcpyHostToDevice);

  double* Vy_cuda;
  double* Vy_cpu = (double*)malloc(nX * (nY + 1) * sizeof(double));
  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY + 1; j++) {
      Vy_cpu[j * nX + i] = 0.0;
    }
  }
  hipMalloc(&Vy_cuda, nX * (nY + 1) * sizeof(double));
  hipMemcpy(Vy_cuda, Vy_cpu, nX * (nY + 1) * sizeof(double), hipMemcpyHostToDevice);

  //std::cout << "Before loop...\n";

  /* ACTION LOOP */
  for (int it = 0; it < NT; it++) {
    ComputeSigma<<<grid, block>>>(Vx_cuda, Vy_cuda, P_cuda, tauXX_cuda, tauYY_cuda, tauXY_cuda, pa_cuda, nX, nY);
    hipDeviceSynchronize();    // wait for compute device to finish
    //std::cout << "After computing sigma...\n";
    ComputeV<<<grid, block>>>(Vx_cuda, Vy_cuda, P_cuda, tauXX_cuda, tauYY_cuda, tauXY_cuda, pa_cuda, nX, nY);
    hipDeviceSynchronize();    // wait for compute device to finish

    hipMemcpy(Vx_cpu, Vx_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Vx on step " << it << " is " << Vx_cpu[nY/2 * (nX + 1) + nX/2] << std::endl;
  }

  /* OUTPUT DATA WRITING */
  hipMemcpy(P_cpu, P_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost);

  FILE* P_filw = fopen("Pc.dat", "wb");
  fwrite(P_cpu, sizeof(double), nX * nY, P_filw);
  fclose(P_filw);

  hipMemcpy(Vx_cpu, Vx_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost);
  FILE* Vx_filw = fopen("Vxc.dat", "wb");
  fwrite(Vx_cpu, sizeof(double), (nX + 1) * nY, Vx_filw);
  fclose(Vx_filw);

  hipMemcpy(tauXY_cpu, tauXY_cuda, (nX - 1) * (nY - 1) * sizeof(double), hipMemcpyDeviceToHost);
  FILE* tauXY_filw = fopen("tauXYc.dat", "wb");
  fwrite(tauXY_cpu, sizeof(double), (nX - 1) * (nY - 1), tauXY_filw);
  fclose(tauXY_filw);

  free(pa_cpu);
  free(P_cpu);
  free(tauXX_cpu);
  free(tauYY_cpu);
  free(tauXY_cpu);
  free(Vx_cpu);
  free(Vy_cpu);

  hipFree(pa_cuda);
  hipFree(P_cuda);
  hipFree(tauXX_cuda);
  hipFree(tauYY_cuda);
  hipFree(tauXY_cuda);
  hipFree(Vx_cuda);
  hipFree(Vy_cuda);

  hipDeviceReset();
  return 0;
}
